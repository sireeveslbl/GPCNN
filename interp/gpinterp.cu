#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "CImg.h"

using namespace std;
using namespace cimg_library;

__global__ void interp(unsigned char * d_src, unsigned char * d_dst, int width, int height)
{
    int pos_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pos_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (pos_x >= width || pos_y >= height)
        return;
	
    uchar3 rgb;
    rgb.x = d_src[pos_y * width + pos_x];
    rgb.y = d_src[(height + pos_y ) * width + pos_x];
    rgb.z = d_src[(height * 2 + pos_y) * width + pos_x];

    unsigned int _gray = (unsigned int)(0.299f*rgb.x + 0.587f*rgb.y + 0.114*rgb.z);
    unsigned char gray = _gray > 255 ? 255 : _gray;

    d_dst[pos_y * width + pos_x] = gray;
}

__device__ float GP( float st[5], float weights[5], float ml[5])
{
	float result = 0.0f; 
	float mle = ml[0]*st[0]+ml[1]*st[1]+ml[2]*st[2]+ml[3]*st[3]+ml[4]*st[4];
	result += mle;
	for(int i = 0; i<5; ++i) result+= weights[i]*(st[i]-mle); 
	return result; 
}


int main()
{
    //load image
    CImg<unsigned char> src("SAGAN.bmp");
    int width = src.width();
    int height = src.height();
    unsigned long size = src.size();

    //create pointer to image
    unsigned char *h_src = src.data();

    CImg<unsigned char> gs(width, height, 1, 1);
    unsigned char *h_gs = gs.data();

    unsigned char *d_src;
    unsigned char *d_gs;

    hipMalloc((void**)&d_src, size);
    hipMalloc((void**)&d_gs, width*height*sizeof(unsigned char));

    hipMemcpy(d_src, h_src, size, hipMemcpyHostToDevice);

    //launch the kernel
    dim3 blkDim (16, 16, 1);
    dim3 grdDim ((width + 15)/16, (height + 15)/16, 1);
    rgb2gray<<<grdDim, blkDim>>>(d_src, d_gs, width, height);

    //wait until kernel finishes
    hipDeviceSynchronize();

    //copy back the result to CPU
    hipMemcpy(h_gs, d_gs, width*height, hipMemcpyDeviceToHost);

    hipFree(d_src);
    hipFree(d_gs);

    CImg<unsigned char> out(h_gs,width,height);
	out.save("GSSAGAN.bmp");
    return 0;
}
